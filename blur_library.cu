// Author: Patrick Wieschollek <mail@patwie.com>
// apply PSF kernel to an image on the GPU
// TODO: create batch-version of run_blur_image

#include <sstream>
#include <iostream>
#include <hip/hip_runtime.h>
#include <hipDNN.h>

#define checkCUDNN(expression)                               \
{                                                            \
  hipdnnStatus_t status = (expression);                       \
  if (status != HIPDNN_STATUS_SUCCESS) {                      \
    std::stringstream strstr;                                \
    strstr << "Error on line " << __LINE__ << ": "           \
    << hipdnnGetErrorString(status) << std::endl;             \
    throw strstr.str();                                      \
  }                                                          \
}

#define checkCUDA(expression)                              \
{                                                          \
  hipError_t error = (expression);                        \
  if (error != hipSuccess) {                              \
    throw std::runtime_error(hipGetErrorString(error));   \
  }                                                        \
}

void run_blur_image
(
  int gpu_id,
  const float *img_h, unsigned int iH, unsigned int iW, unsigned int iC,
  const float *psf_h, unsigned int pH, unsigned int pW,
  float **output_d) {

  hipSetDevice(gpu_id);

  const int image_bytes = iC * iH * iW * sizeof(float);
  const int psf_bytes = pH * pW * sizeof(float);

  // copy to device memory
  float *img_d;
  checkCUDA(hipMalloc(&img_d, image_bytes));
  checkCUDA(hipMemcpy(img_d, img_h, image_bytes, hipMemcpyHostToDevice));

  float *psf_d;
  checkCUDA(hipMalloc(&psf_d, psf_bytes));
  checkCUDA(hipMemcpy(psf_d, psf_h, psf_bytes, hipMemcpyHostToDevice));

  checkCUDA(hipMalloc(&*output_d, image_bytes));
  checkCUDA(hipMemset(*output_d, 0, image_bytes));


  hipdnnHandle_t cudnn;
  hipdnnCreate(&cudnn);

  hipdnnTensorDescriptor_t input_descriptor;
  checkCUDNN(hipdnnCreateTensorDescriptor(&input_descriptor));
  checkCUDNN(hipdnnSetTensor4dDescriptor(
               input_descriptor,
               HIPDNN_TENSOR_NCHW, HIPDNN_DATA_FLOAT, 1, 1, iH, iW));

  hipdnnFilterDescriptor_t kernel_descriptor;
  checkCUDNN(hipdnnCreateFilterDescriptor(&kernel_descriptor));
  checkCUDNN(hipdnnSetFilter4dDescriptor(
               kernel_descriptor,
               HIPDNN_DATA_FLOAT, HIPDNN_TENSOR_NCHW, 1, 1, pH, pW));

  hipdnnConvolutionDescriptor_t convolution_descriptor;
  checkCUDNN(hipdnnCreateConvolutionDescriptor(&convolution_descriptor));
  checkCUDNN(hipdnnSetConvolution2dDescriptor(
               convolution_descriptor,
               (pH - 1) / 2, (pW - 1) / 2, 1, 1, 1, 1,
               HIPDNN_CROSS_CORRELATION, HIPDNN_DATA_FLOAT));

  hipdnnTensorDescriptor_t output_descriptor;
  checkCUDNN(hipdnnCreateTensorDescriptor(&output_descriptor));
  checkCUDNN(hipdnnSetTensor4dDescriptor(
               output_descriptor,
               HIPDNN_TENSOR_NCHW, HIPDNN_DATA_FLOAT, 1, 1, iH, iW));

  hipdnnConvolutionFwdAlgo_t convolution_algorithm;
  checkCUDNN(hipdnnGetConvolutionForwardAlgorithm(
               cudnn,
               input_descriptor,
               kernel_descriptor,
               convolution_descriptor,
               output_descriptor,
               HIPDNN_CONVOLUTION_FWD_PREFER_FASTEST,
               0, &convolution_algorithm));

  size_t workspace_bytes{0};
  checkCUDNN(hipdnnGetConvolutionForwardWorkspaceSize(
               cudnn,
               input_descriptor,
               kernel_descriptor,
               convolution_descriptor,
               output_descriptor,
               convolution_algorithm,
               &workspace_bytes));

  void* d_workspace{nullptr};
  hipMalloc(&d_workspace, workspace_bytes);

  const float alpha = 1.0f, beta = 0.0f;
  // no NVIDIA-cudnn depthwise primitive?
  for (int i = 0; i < iC; ++i) {
    checkCUDNN(hipdnnConvolutionForward(cudnn,
                                       &alpha,
                                       input_descriptor,
                                       img_d + i * iH * iW,
                                       kernel_descriptor,
                                       psf_d,
                                       convolution_descriptor,
                                       convolution_algorithm,
                                       d_workspace,
                                       workspace_bytes,
                                       &beta,
                                       output_descriptor,
                                       *output_d + i * iH * iW));
  }

  checkCUDA(hipFree(psf_d));
  checkCUDA(hipFree(img_d));
  checkCUDA(hipFree(d_workspace));
  hipdnnDestroyTensorDescriptor(input_descriptor);
  hipdnnDestroyTensorDescriptor(output_descriptor);
  hipdnnDestroyFilterDescriptor(kernel_descriptor);
  hipdnnDestroyConvolutionDescriptor(convolution_descriptor);

  hipdnnDestroy(cudnn);
}


